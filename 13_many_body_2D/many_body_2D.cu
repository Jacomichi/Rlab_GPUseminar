#include "hip/hip_runtime.h"
#include <iomanip>
#include <limits>
#include <algorithm>
#include <cassert>

#include "thrust_all.cuh"
#include "./utility/timer.cuh"
#include "./utility/random.cuh"

#include "./src/particle.cuh"
#include "./src/mesh.cuh"
#include "./src/list.cuh"
#include "./src/file.cuh"
#include "./src/md_host.cu"

int main(){
    //Simulation parameter
    int N = 1<<12;
    double rho = 0.8;
    double temperature = 1.0;
    double friction = 1.0;
    double large_sigma = 1.4;

    Atoms atoms(N,rho,temperature,friction,large_sigma);

    //simulation setting
    double dt = 0.002;
    int tau = 1.0/dt;
    int max_steps = 30000*tau;
    int stabilization_steps = 100*tau;
    int equilibration_steps = 1000*tau;
    Setting setting(N,rho,dt);
    std::cout << "L : " << setting.L << '\n';

    auto gen = createRandGenerator();
    atoms.set_diameter();
    atoms.random_uniform(gen,setting.L);
    //atoms.create_square_lattice(setting.L);
    atoms.fillVelo(0.0);


    std::cout << "finish setup" << '\n';
    //Output file setting
    int sampling_span = 100*tau;
    std::string dir_path_conf = create_outfile_conf_binary(atoms);
    //std::string dir_path_energy = create_outfile_energy_binary(atoms);
    Output outfile(dir_path_conf);
    //Output energy(dir_path_energy);

    //Timer setting
    cudaTimer timer;


    {//mesh setting
    double mesh_size = 2.3;
    int mesh_num = ceil(setting.L/mesh_size);
    assert(mesh_num > 3);
    Mesh mesh(N,mesh_num,setting.L);

    //List setting
    int list_size = 25;
    double list_cutoff = 2.0;
    unsigned int refresh_span = 100;//100回に1回更新
    List list(N,list_size,list_cutoff,refresh_span);
    std::cout << "start stabilization" << '\n';
    for(int t = 0; t < stabilization_steps; ++t){
        if( (t % list.refresh_span) == 0){
            h_update_verlet_list(atoms,list,mesh,setting);
        }
        // if( (t % (10*tau)) == 0){
        //     std::cout << "t : " << t/tau << '\n';
        //     //outfile.write_configuration(atoms);
        // }
        quench(atoms,list,mesh,setting);
    }
}


    double mesh_size = 3.5;
    int mesh_num = ceil(setting.L/mesh_size);
    assert(mesh_num > 3);
    Mesh mesh(N,mesh_num,setting.L);

    //List setting
    int list_size = 40;
    double list_cutoff = 3.5;
    unsigned int refresh_span = 100;//100回に1回更新
    List list(N,list_size,list_cutoff,refresh_span);



    std::cout << "start equlibration" << '\n';
    for(int t = 0; t < equilibration_steps; ++t){
        if( (t % list.refresh_span) == 0){
            h_update_verlet_list(atoms,list,mesh,setting);
        }
        EoM(atoms,list,mesh,setting,gen);
    }
    std::cout << "finish equilibration" << '\n';

    std::cout << "simulation start" << '\n';
    double ave_kin = 0.0;
    double ave_pot = 0.0;
    timer.start_record();

    for(int t = 0; t < max_steps; ++t){
        if( (t % list.refresh_span) == 0){
            //h_create_list_full_search(atoms,list,setting);
            h_update_verlet_list(atoms,list,mesh,setting);
            // ave_kin = 0.0;
            // ave_kin = atoms.average_kinetic();
            // ave_pot = h_calc_potential_energy(atoms,setting,list);
            // std::cout << "step : " << t << " kinetic energy : " << ave_kin << " average potential : " << ave_pot << '\n';
            //std::cout << "step : " << t << " kinetic energy : " << ave_kin  << '\n';
            //outfile.write_configuration(atoms);
        }
        if( (t % sampling_span) == 0){
            // ave_kin = 0.0;
            // ave_kin = atoms.average_kinetic();
            // ave_pot = h_calc_potential_energy(atoms,setting,list);
            // std::cout << "step : " << t << " kinetic energy : " << ave_kin << " average potential : " << ave_pot << '\n';
            //std::cout << "step : " << t << " kinetic energy : " << ave_kin  << '\n';
            outfile.write_configuration(atoms);
        }
        EoM(atoms,list,mesh,setting,gen);
    }
    std::cout << "finish simulation" << '\n';


    timer.stop_record();
    timer.print_result();

    hiprandDestroyGenerator(gen);

    return 0;
}
