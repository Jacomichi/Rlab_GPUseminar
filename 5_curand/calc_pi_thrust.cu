#include <random>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "thrust_all.cuh"

hiprandGenerator_t createRandGenerator(){
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    std::random_device rnd;
    hiprandSetPseudoRandomGeneratorSeed(gen,rnd());
    return gen;
}

struct is_in{
    typedef thrust::tuple<float,float> tfloat2;
    __host__ __device__
    float operator()(tfloat2 r){
        float _x = thrust::get<0>(r);
        float _y = thrust::get<1>(r);
        return (_x * _x + _y * _y < 1.0f) ? 1.0f : 0.0f;
    }
};

int main(){
    constexpr unsigned int N = 1<<30;

    hipEvent_t start, stop;
    float elapse;

    // initialize time
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record initial time
    hipEventRecord(start);


    thrust::device_vector<float> x(N);
    thrust::device_vector<float> y(N);

    auto gen = createRandGenerator();

    hiprandGenerateUniform(gen,thrust::raw_pointer_cast(&x[0]),N);
    hiprandGenerateUniform(gen,thrust::raw_pointer_cast(&y[0]),N);
    hipDeviceSynchronize();

    /*
    auto is_in_circle = []__device__(float _x,float _y){
        return (_x * _x + _y * _y < 1.0f) ? 1.0f : 0.0f;
    };
    thrust::transform(x.begin(),x.end(),y.begin(),result.begin(),is_in_circle);
    double pi = thrust::reduce(result.begin(),result.end(),0.0f,thrust::plus<float>());
    */

    auto first = thrust::make_zip_iterator(thrust::make_tuple(x.begin(),y.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(x.end(),y.end()));
    double pi =thrust::transform_reduce(first,last,is_in(),0.0f,thrust::plus<float>());

    std::cout << 4.0 * pi/N << '\n';
    hipEventRecord(stop);

    //wait until all events complete
    hipEventSynchronize(stop);

    // calc
    hipEventElapsedTime(&elapse, start, stop);
    std::cout<<"gpu :"<<elapse<<"ms"<<std::endl;
    // 終了処理
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hiprandDestroyGenerator(gen);

    return 0;
}
