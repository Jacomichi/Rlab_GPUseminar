#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <random>
#include <iostream>
#include <algorithm>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "thrust_all.cuh"

//hiprandStateの初期化
__global__ void setCurand(unsigned long long seed, hiprandState *state){
    uint i_global = threadIdx.x + blockIdx.x*blockDim.x;
    hiprand_init(seed, i_global, 0, &state[i_global]);
}

struct execSetting{
    dim3 grid;
    dim3 block;
    execSetting(dim3 _grid,dim3 _block){
        grid = _grid;
        block = block;
    }
    execSetting(int gridsize,int blocksize){
        dim3 _grid(gridsize);
        grid = _grid;
        dim3 _block(blocksize);
        block = _block;
    }
};

void setRand(hiprandState *state,execSetting set){
    std::random_device _rnd;
    setCurand<<<set.grid,set.block>>>(_rnd(), state);
}

template <typename T>
std::size_t check_size(int N){
    return N * sizeof(T);
}

//一様乱数を返す
__global__ void genrand_kernel(float *result, hiprandState *state){

    uint i_global = threadIdx.x + blockIdx.x*blockDim.x;
    result[i_global] = hiprand_normal(&state[i_global]);
}

__global__ void calc_pi(float *result, hiprandState *state){
    auto idx = threadIdx.x + blockIdx.x*blockDim.x;
    auto rnd_x = hiprand_uniform(&state[idx]);
    auto rnd_y = hiprand_uniform(&state[idx]);
    result[idx] = (rnd_x * rnd_x + rnd_y * rnd_y < 1.0f) ? 1.0f : 0.0f;
}


int main(){
    constexpr unsigned int N = 1<<24;
    constexpr unsigned int num_Blocks = 1<<14;
    unsigned int threads_per_blocks = std::min(std::ceil(static_cast<double>(N)/num_Blocks),1024.0);

    hipEvent_t start, stop;
    float elapse;

    // initialize time
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // record initial time
    hipEventRecord(start);

    thrust::device_vector<float> result(N);
    thrust::device_vector<hiprandState> state(N);
    thrust::host_vector<float> hresult(N);
    float *result_ptr = thrust::raw_pointer_cast(&result[0]);
    hiprandState *state_ptr = thrust::raw_pointer_cast(&state[0]);

    execSetting set(num_Blocks,threads_per_blocks);

    setRand(state_ptr, set);

    calc_pi<<<set.grid,set.block>>>(result_ptr,state_ptr);
    double pi = thrust::reduce(result.begin(),result.end(),0.0f,thrust::plus<float>());

    std::cout << 4.0/N*pi << '\n';
    hipEventRecord(stop);

    //wait until all events complete
    hipEventSynchronize(stop);

    // calc
    hipEventElapsedTime(&elapse, start, stop);
    std::cout<<"gpu :"<<elapse<<"ms"<<std::endl;
    // 終了処理
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
