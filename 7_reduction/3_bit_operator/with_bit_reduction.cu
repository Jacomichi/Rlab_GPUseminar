#include <random>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "thrust_all.cuh"
#include "reduction.cuh"

hiprandGenerator_t createRandGenerator(){
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    std::random_device rnd;
    hiprandSetPseudoRandomGeneratorSeed(gen,rnd());
    return gen;
}


int main(){
    constexpr unsigned int N = 1e7;
    constexpr int num_threads = 256;
    constexpr int trial_number = 100;

    hipEvent_t start, stop;
    float elapse;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    thrust::device_vector<double> input(N);
    thrust::device_vector<double> output(N);

    auto gen = createRandGenerator();

    hiprandGenerateNormalDouble(gen,thrust::raw_pointer_cast(&input[0]),N,0.0,1.0);
    hipDeviceSynchronize();

    auto output_ptr = thrust::raw_pointer_cast(&output[0]);
    //trial
    for(int i = 0; i < trial_number;++i){
        thrust::copy(input.begin(),input.end(),output.begin());
        reduction(output_ptr,output_ptr, num_threads, N);
        hipDeviceSynchronize();
    }
    std::cout << output[0]/static_cast<double>(N)  << '\n';


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapse, start, stop);
    std::cout<<"gpu :"<<elapse / static_cast<double>(trial_number) <<"ms"<<std::endl;
    float bandwidth = N * sizeof(double) / elapse / 1e6;
    std::cout <<  "bandwidth= " << bandwidth << "GB/s\n";
    // 終了処理
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hiprandDestroyGenerator(gen);

    return 0;
}
