#include <hip/hip_runtime.h>
#include "cuda_utility.cuh"
#include <cassert>

#ifndef _CUDA_UTIL_FUNC_

void allocateArray(void **devPtr, size_t size)
{
	CudaSafeCall(hipMalloc(devPtr, size));
}

template < typename T >
void allocateUM(T *devPtr, size_t size)
{
	CudaSafeCall(hipMallocManaged(devPtr, size));
}

void freeArray(void *devPtr)
{
	CudaSafeCall(hipFree(devPtr));
}

void threadSync()
{
	CudaSafeCall(hipDeviceSynchronize());
}

void copyD2H(void* dst, void* src, size_t memSize) {
	CudaSafeCall(hipMemcpy(dst, src, memSize, hipMemcpyDeviceToHost));
}

void copyH2D(void* dst, void* src, size_t memSize) {
	CudaSafeCall(hipMemcpy(dst, src, memSize, hipMemcpyHostToDevice));
}


#define copyH2S(symbol, src, memSize) \
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(symbol), src, memSize))


#endif
